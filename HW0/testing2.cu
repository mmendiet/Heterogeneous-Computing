
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>

#include <iostream>
#include <fstream>
#include <chrono>

//#define N 1000
//#define M 512
//nvcc testing.cu -o test
//


__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}

void cpuAdd(int *a, int *b, int *c, int n) {
    for(int i=0; i<n; ++i) {
        c[i] = a[i] + b[i];
    }
}

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(int argc, char* argv[]) {
    int N = atoi(argv[1]);
    int M = atoi(argv[2]);
    int *a, *b, *c;

    // device copies of a, b, c
    int size = N * sizeof(int);

    // Setup input values
    a = (int*)malloc(size); random_ints(a, N);
    b = (int*)malloc(size); random_ints(b, N);
    c = (int*)malloc(size);  
    if (strcmp(argv[3],"gpu")==0) {
        // host copies of a, b, c
        int *d_a, *d_b, *d_c;
        // Allocate space for device copies of a, b, c

        float time;
        hipEvent_t start, stop;
        
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord( start, 0 );

        hipMalloc((void **)&d_a, size);
        hipMalloc((void **)&d_b, size);
        hipMalloc((void **)&d_c, size);

        // Copy inputs to device
        hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
        hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
        // Launch add() kernel on GPU

        add<<<(N + M-1) / M,M>>>(d_a, d_b, d_c, N);

    
        // Copy result back to host
        hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
        hipEventRecord( stop, 0 );
        hipEventSynchronize( stop );

        hipEventElapsedTime( &time, start, stop );
        hipEventDestroy( start );
        hipEventDestroy( stop );

        hipFree(d_a); hipFree(d_b); hipFree(d_c);
        float nanosec = (time)*1000000;
        std::cout << "N: " << N << "   M: " << M << "   GPU time: " << nanosec << "ns" << std::endl;
    }

    else {
        auto t1 = std::chrono::high_resolution_clock::now();
        cpuAdd(a,b,c, N);
        auto t2 = std::chrono::high_resolution_clock::now();
        auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>( t2 - t1 ).count();
        std::cout << "N: " << N << "   M: " << M << "   CPU time: " << duration << "ns" << std::endl;
        //time = 100.0;
    }
    //printf("a[0]: %i, b[0]: %i, c[0]: %i\nGPU Time: %f\n", a[0], b[0], c[0], time);
    //printf("N: %i   M: %i   Time: %f\n", N, M, time);
    // Cleanup
    free(a); free(b); free(c);

    return 0;
}
